﻿#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <math.h>

#ifndef __CUDACC__
#define __CUDACC__
#include <device_functions.h>
#endif
#include <string>

void readMatrixAndVectorFromFile();
void readFloatRowFormFile(FILE* fp, int size_of_row, float** data);
void readIntRowFromFile(FILE* fp, int size_of_row, int** data);
int calculate_grid_dimension(int matrix_dimension);
__global__ void jacobi(int matrix_dimension, int* prefix_array, int* rows_coo, int offset_array_size, float* data_ell, int* cols_ell, int size_of_ell_row, float* x, float* y, int data_ell_size, float* vector, int data_coo_size, int size_of_coo_row, float* data_coo, int* cols_coo);
__global__ void offset(int* offset_array, int* rows_coo, int data_coo_size, int size_of_coo_row);
__global__ void init_result_vector(int matrix_dimension, float* vector);

const char* matrix_file_name = "matrix_ell_coo_15.csv";
int matrix_dimension = 0;

// Daten der Matrix im ELL Format
int data_ell_size = 0;
int cols_ell_size = 0;
int size_of_ell_row = 0;
float* data_ell = NULL;
int* cols_ell = NULL;

// Daten der Matrix im COO Format
int data_coo_size = 0;
int size_of_coo_row = 0;
float* data_coo = NULL;
int* rows_coo = NULL;
int* cols_coo = NULL;

// Der Ergebnisvektor
float* vector = NULL;

// Prefix array
int* offset_array = NULL;

// Intermitted result vectors
float* x = NULL;
float* y = NULL;

int main()
{
    // Lese Matrix und Vektor aus der Eingabedatei
    readMatrixAndVectorFromFile();

    // Berechne wie groß die Grid Dimension sein muss (bei BlockDim 1024)
    int grid_dimension = calculate_grid_dimension(matrix_dimension);

    // Initialisiere den Ergebnisvektor
    hipMallocManaged(&x, matrix_dimension);
    init_result_vector<<<grid_dimension, 1024 >>>(matrix_dimension, x);

    // Vektor für das Zwischenergebnis
    hipMallocManaged(&y, matrix_dimension);

    // Initializiere und berechne Offset Array
    hipMallocManaged(&offset_array, data_coo_size / size_of_coo_row);
    offset<<<1,1024>>> (offset_array, rows_coo, data_coo_size, size_of_coo_row);
    hipDeviceSynchronize();

    // Starte die Jacobi Iterationen
    for (int k = 0; k < 1; k++) {
        jacobi<<<grid_dimension, 1024>>> (matrix_dimension, offset_array, rows_coo, (data_coo_size/ size_of_coo_row), data_ell, cols_ell, size_of_ell_row, x, y, data_ell_size, vector, data_coo_size, size_of_coo_row, data_coo, cols_coo);
        hipDeviceSynchronize();
        // check Iteration
    }

    // evaluate result

    hipDeviceSynchronize();

    hipFree(data_ell);
    hipFree(cols_ell);
    hipFree(data_coo);
    hipFree(rows_coo);
    hipFree(cols_coo);
    hipFree(vector);

    return 0;
}

__global__ void init_result_vector(int matrix_dimension, float* vector) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < matrix_dimension) {
        vector[idx] = 0.0;
    }
}

__global__ void offset(int* offset_array, int* rows_coo, int data_coo_size, int size_of_coo_row) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < (data_coo_size / size_of_coo_row)) {
        printf("IDX:%d = %d\n", idx, rows_coo[idx * size_of_coo_row]);
        offset_array[idx] = rows_coo[idx * size_of_coo_row];
    }
}

__global__ void jacobi(int matrix_dimension, int* offset_array, int* rows_coo, int offset_array_size, float* data_ell, int* cols_ell, int size_of_ell_row, float* x, float* y, int data_ell_size, float* vector, int data_coo_size, int size_of_coo_row, float* data_coo, int* cols_coo)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < matrix_dimension) {
        printf("This is thread number %d\n", idx);

        int row_offset = 0;
        int i;
        bool is_coo_row = false;
        int index_of_diagonal_element;

        for (i=0; i < offset_array_size; i++) {
            if (offset_array[i] == idx) {
                // Zeile ist eine COO Zeile
                is_coo_row = true;
                break;
            }
            else {
                // Zeile ist eine ELL Zeile
                // Berechne row_offset (Wie viele COO Zeilen gab es bis idx)
                if (offset_array[i] < idx) {
                    row_offset++;
                }
                else {
                    break;
                }
            }
        }
        if (is_coo_row) {
            printf("IDX:%d is a COO row\n", idx);          
            for (int m = i*size_of_coo_row; m < size_of_coo_row; m++) {

                if (idx != cols_coo[m]) {
                    y[idx] -= data_coo[m] * x[cols_coo[m]];
                }
                else {
                    y[idx] += vector[idx];
                    index_of_diagonal_element = m;
                }
            }
            y[idx] = y[idx] / data_coo[index_of_diagonal_element];
        }
        else {
            printf("IDX:%d is a ELL row with row_offset %d\n", idx, row_offset);
            for (int i = idx - row_offset; i < data_ell_size; i = i + size_of_ell_row) {

                if (idx != cols_ell[i]) {
                    y[idx] -= data_ell[i] * x[cols_ell[i]];
                }
                else {
                    y[idx] += vector[idx];
                    index_of_diagonal_element = i;
                }
            }
            y[idx] = y[idx] / data_ell[index_of_diagonal_element];
        }
    }
}

int calculate_grid_dimension(int dimension) {
    return (int)ceil(dimension / static_cast<double>(1024));
}

void readMatrixAndVectorFromFile()
{
    FILE* fp;
    fp = fopen(matrix_file_name, "r");
    fscanf(fp, "%d,%d", &matrix_dimension, &size_of_ell_row);

    // Lese ELL Format
    fscanf(fp, "%d,%d", &data_ell_size, &cols_ell_size);

    hipMallocManaged(&data_ell, data_ell_size * sizeof(float));
    hipMallocManaged(&cols_ell,cols_ell_size * sizeof(int));
    hipMallocManaged(&vector, matrix_dimension * sizeof(float));
    readFloatRowFormFile(fp, data_ell_size, &data_ell);
    readIntRowFromFile(fp, cols_ell_size, &cols_ell);

    printf("Dimension: %d\nELL Data Size: %d\nELL Cols Size: %d\n", matrix_dimension, data_ell_size, cols_ell_size);
    printf("ELL Row Size: %d\n", size_of_ell_row);
    printf("Data ELL: ");
    for (int i = 0; i < data_ell_size; i++) {
        printf("%f ", data_ell[i]);
    }
    printf("\n");
    printf("Cols ELL: ");
    for (int i = 0; i < cols_ell_size; i++) {
        printf("%d ", cols_ell[i]);
    }
    printf("\n");
    
    // Lese COO Format
    fscanf(fp, "%d,%d", &data_coo_size, &size_of_coo_row);
    hipMallocManaged(&data_coo, data_coo_size * sizeof(float));
    hipMallocManaged(&rows_coo, data_coo_size * sizeof(int));
    hipMallocManaged(&cols_coo, data_coo_size * sizeof(int));
    readFloatRowFormFile(fp, data_coo_size, &data_coo);
    readIntRowFromFile(fp, data_coo_size, &rows_coo);
    readIntRowFromFile(fp, data_coo_size, &cols_coo);

    printf("COO Data Size: %d\n", data_coo_size);
    printf("COO Row Size: %d\n", size_of_coo_row);
    printf("Data COO: ");
    for (int i = 0; i < data_coo_size; i++) {
        printf("%f ", data_coo[i]);
    }
    printf("\n");
    printf("Rows COO: ");
    for (int i = 0; i < data_coo_size; i++) {
        printf("%d ", rows_coo[i]);
    }
    printf("\n");

    printf("Cols COO: ");
    for (int i = 0; i < data_coo_size; i++) {
        printf("%d ", cols_coo[i]);
    }
    printf("\n");

    // Lese Vektor
    readFloatRowFormFile(fp, matrix_dimension, &vector);
    printf("Vector: ");
    for (int i = 0; i < matrix_dimension; i++) {
        printf("%f ", vector[i]);
    }
    printf("\n");

    fclose(fp);
    return;
}

void readFloatRowFormFile(FILE* fp, int size_of_row, float** data) {
    int valuesRead = 0;
    char* row = (char*)malloc((size_of_row * 8 + size_of_row) * sizeof(char));
    
    fscanf(fp, "%s\n", row);
    char* ptr = strtok(row, ",");
    
    while (ptr != NULL)
    {
        (*data)[valuesRead] = atof(ptr);
        valuesRead++;
        ptr = strtok(NULL, ",");
    }

    free(row);
    return;
}

void readIntRowFromFile(FILE* fp, int size_of_row, int** data) {
    int valuesRead = 0;
    char* row = (char*)malloc((size_of_row * 6 + size_of_row) * sizeof(char));

    fscanf(fp, "%s\n", row);
    char* ptr = strtok(row, ",");
   
    while (ptr != NULL)
    {
        (*data)[valuesRead] = atoi(ptr);
        valuesRead++;
        ptr = strtok(NULL, ",");
    }
    free(row);
    return;
}